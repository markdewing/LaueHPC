#include "hip/hip_runtime.h"

#include "solve_cuda.h"
#include "perf_info.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdexcept>


void solve_cuda_QR(int nrow, int ncol, double* A_ptr, double* b_ptr, double* result_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    double* dA;
    err = hipMalloc((void **)&dA, sizeof(double) * nrow * ncol);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    double *dtau;
    err = hipMalloc((void **)&dtau, sizeof(double) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    double* db;
    err = hipMalloc((void **)&db, sizeof(double) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");


    err = hipMemcpy(dA, A_ptr, sizeof(double)*nrow*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db, b_ptr, sizeof(double)*nrow, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");

    int lwork_geqrf;
    hipsolverDnDgeqrf_bufferSize(cusolverH, nrow, ncol, dA, nrow, &lwork_geqrf);

    int lwork_ormqr;
    hipsolverDnDormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA, nrow, dtau, db, nrow, &lwork_ormqr);

    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    double *dwork;
    err = hipMalloc((void **)&dwork, sizeof(double) * lwork);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


    hipsolverStatus_t serr;
    serr = hipsolverDnDgeqrf(cusolverH, nrow, ncol, dA, nrow, dtau, dwork, lwork, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
     std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

    serr = hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork, lwork, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
        std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

    double one(1.0);
    hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

    hipMemcpy(result_ptr, db, sizeof(double)*ncol, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipFree(dA);
    hipFree(dtau);
    hipFree(db);
    hipFree(dinfo);
    hipFree(dwork);
}


void solve_cuda_SVD(int nrow, int ncol, double* A_ptr, double* b_ptr, double* result_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    double* dA;
    err = hipMalloc((void **)&dA, sizeof(double) * nrow * ncol);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    double* db;
    err = hipMalloc((void **)&db, sizeof(double) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");

    double* S = new double[min_mn];

    double* dS;
    err = hipMalloc((void **)&dS, sizeof(double) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dS");

    double* dU;
    err = hipMalloc((void **)&dU, sizeof(double) * nrow * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dU");

    double* dVT;
    err = hipMalloc((void **)&dVT, sizeof(double) * ncol * ncol);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dVT");



    err = hipMemcpy(dA, A_ptr, sizeof(double)*nrow*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db, b_ptr, sizeof(double)*nrow, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");


    int lwork_gesvd;
    hipsolverDnDgesvd_bufferSize(cusolverH, nrow, ncol, &lwork_gesvd);

    int lwork = lwork_gesvd;

    printf("lwork size = %d\n",lwork);
    double *dwork;
    err = hipMalloc((void **)&dwork, sizeof(double) * lwork);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");

    char jobu('A');
    char jobvt('A');

    hipsolverStatus_t serr;
    serr = hipsolverDnDgesvd(cusolverH, jobu, jobvt, nrow, ncol, dA, nrow, dS, dU, nrow, dVT, ncol, dwork, lwork, nullptr, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgesvd failed"  + std::to_string(serr));

    err = hipMemcpy(S, dS, sizeof(double)*min_mn, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy S_ptr");

    for (int i = 0; i < ncol; i++) {
        S[i] = 1.0/S[i];
    }

    double* tmp_ptr  = new double[nrow];
    double* dtmp;
    err = hipMalloc((void **)&dtmp, sizeof(double) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtmp");

    // u.T * b
    double one(1.0);
    double zero(0.0);
    int incx(1);
    hipblasDgemv(cublasH, HIPBLAS_OP_T, nrow, nrow, &one, dU, nrow, db, incx, &zero, dtmp, incx);

    err = hipMemcpy(tmp_ptr, dtmp, sizeof(double)*ncol, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy tmp_ptr to host");

    // S^-1 * (u.T * b)
    for (int i = 0; i < ncol; i++) {
        tmp_ptr[i] *= S[i];
    }

    err = hipMemcpy(dtmp, tmp_ptr, sizeof(double)*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy tmp_ptr to device");

    // v.T * (S^-1 * (u.T * b))
    hipblasDgemv(cublasH, HIPBLAS_OP_T, ncol, ncol, &one, dVT, ncol, dtmp, incx, &zero, db, incx);

    hipMemcpy(result_ptr, db, sizeof(double)*ncol, hipMemcpyDeviceToHost);

    hipFree(dtmp);
    hipFree(dwork);
    hipFree(dVT);
    hipFree(dU);
    hipFree(dS);
    hipFree(db);
    hipFree(dA);

    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(cublasH);

    delete[] S;
    delete[] tmp_ptr;
}


void solve_batch_cuda_QR(int nrow, int ncol, int nbatch, double* A_batch_ptr, double* b_batch_ptr, double* result_batch_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    int nstream = nbatch;
    hipStream_t* streams = new hipStream_t[nstream];
    hipsolverHandle_t* cusolverH = new hipsolverHandle_t[nstream];
    hipblasHandle_t* cublasH = new hipblasHandle_t[nstream];
    for (int i = 0; i < nstream; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        //hipStreamCreate(&streams[i]);
        hipsolverDnCreate(&cusolverH[i]);
        hipsolverSetStream(cusolverH[i], streams[i]);
        hipblasCreate(&cublasH[i]);
        hipblasSetStream(cublasH[i], streams[i]);
    }

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    double* dA_batch;
    err = hipMalloc((void **)&dA_batch, sizeof(double) * nrow * ncol * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    double *dtau;
    err = hipMalloc((void **)&dtau, sizeof(double) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    double* db_batch;
    err = hipMalloc((void **)&db_batch, sizeof(double) * nrow * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");

#if 0
    err = hipMemcpy(dA_batch, A_batch_ptr, sizeof(double)*nrow*ncol*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db_batch, b_batch_ptr, sizeof(double)*nrow*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");
#endif
#if 1
    for (int ib = 0; ib < nbatch; ib++)  {
        err = hipMemcpyAsync(dA_batch + ib*nrow*ncol, A_batch_ptr, sizeof(double)*nrow*ncol, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy dA");

        err = hipMemcpyAsync(db_batch + ib*nrow , b_batch_ptr, sizeof(double)*nrow, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy db");
    }
#endif


    int lwork_geqrf;
    hipsolverDnDgeqrf_bufferSize(cusolverH[0], nrow, ncol, dA_batch, nrow, &lwork_geqrf);

    int lwork_ormqr;
    hipsolverDnDormqr_bufferSize(cusolverH[0], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA_batch, nrow, dtau, db_batch, nrow, &lwork_ormqr);

    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    double *dwork;
    err = hipMalloc((void **)&dwork, sizeof(double) * lwork * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


// Try launching streams in multiple threads
#pragma omp parallel for
    for (int ib = 0; ib < nbatch; ib++) {
        double *dA = dA_batch + ib*nrow*ncol;
        double *db = db_batch + ib*nrow;
        hipsolverStatus_t serr;

        serr = hipsolverDnDgeqrf(cusolverH[ib], nrow, ncol, dA, nrow, dtau, dwork + lwork*ib, lwork, dinfo);
        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        // std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

        serr = hipsolverDnDormqr(cusolverH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork + lwork*ib, lwork, dinfo);
        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        //    std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

        double one(1.0);
        hipblasDtrsm(cublasH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

        hipMemcpyAsync(result_batch_ptr + ib*ncol, db, sizeof(double)*ncol, hipMemcpyDeviceToHost, streams[ib]);
    }


    //hipMemcpy(result_batch_ptr, db_batch, sizeof(double)*nrow*nbatch, hipMemcpyDeviceToHost);
    //for (int i = 0; i < nstream; i++)
    //    hipStreamSynchronize(streams[i]);

    hipDeviceSynchronize();



    hipFree(dA_batch);
    hipFree(dtau);
    hipFree(db_batch);
    hipFree(dinfo);
    hipFree(dwork);
}
