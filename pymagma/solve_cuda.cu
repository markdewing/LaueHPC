#include "hip/hip_runtime.h"

#include "solve_cuda.h"
#include "perf_info.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdexcept>


void solve_cuda_QR(int nrow, int ncol, double* A_ptr, double* b_ptr, double* result_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    double* dA;
    err = hipMalloc((void **)&dA, sizeof(double) * nrow * ncol);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    double *dtau;
    err = hipMalloc((void **)&dtau, sizeof(double) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    double* db;
    err = hipMalloc((void **)&db, sizeof(double) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");


    err = hipMemcpy(dA, A_ptr, sizeof(double)*nrow*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db, b_ptr, sizeof(double)*nrow, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");

    int lwork_geqrf;
    hipsolverDnDgeqrf_bufferSize(cusolverH, nrow, ncol, dA, nrow, &lwork_geqrf);

    int lwork_ormqr;
    hipsolverDnDormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA, nrow, dtau, db, nrow, &lwork_ormqr);

    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    double *dwork;
    err = hipMalloc((void **)&dwork, sizeof(double) * lwork);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


    hipsolverStatus_t serr;
    serr = hipsolverDnDgeqrf(cusolverH, nrow, ncol, dA, nrow, dtau, dwork, lwork, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
     std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

    serr = hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork, lwork, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
        std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

    double one(1.0);
    hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

    hipMemcpy(result_ptr, db, sizeof(double)*ncol, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipFree(dA);
    hipFree(dtau);
    hipFree(db);
    hipFree(dinfo);
    hipFree(dwork);
}

void solve_batch_cuda_QR(int nrow, int ncol, int nbatch, double* A_batch_ptr, double* b_batch_ptr, double* result_batch_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    int nstream = nbatch;
    hipStream_t* streams = new hipStream_t[nstream];
    hipsolverHandle_t* cusolverH = new hipsolverHandle_t[nstream];
    hipblasHandle_t* cublasH = new hipblasHandle_t[nstream];
    for (int i = 0; i < nstream; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        //hipStreamCreate(&streams[i]);
        hipsolverDnCreate(&cusolverH[i]);
        hipsolverSetStream(cusolverH[i], streams[i]);
        hipblasCreate(&cublasH[i]);
        hipblasSetStream(cublasH[i], streams[i]);
    }

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    double* dA_batch;
    err = hipMalloc((void **)&dA_batch, sizeof(double) * nrow * ncol * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    double *dtau;
    err = hipMalloc((void **)&dtau, sizeof(double) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    double* db_batch;
    err = hipMalloc((void **)&db_batch, sizeof(double) * nrow * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");

#if 0
    err = hipMemcpy(dA_batch, A_batch_ptr, sizeof(double)*nrow*ncol*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db_batch, b_batch_ptr, sizeof(double)*nrow*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");
#endif
#if 1
    for (int ib = 0; ib < nbatch; ib++)  {
        err = hipMemcpyAsync(dA_batch + ib*nrow*ncol, A_batch_ptr, sizeof(double)*nrow*ncol, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy dA");

        err = hipMemcpyAsync(db_batch + ib*nrow , b_batch_ptr, sizeof(double)*nrow, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy db");
    }
#endif


    int lwork_geqrf;
    hipsolverDnDgeqrf_bufferSize(cusolverH[0], nrow, ncol, dA_batch, nrow, &lwork_geqrf);

    int lwork_ormqr;
    hipsolverDnDormqr_bufferSize(cusolverH[0], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA_batch, nrow, dtau, db_batch, nrow, &lwork_ormqr);

    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    double *dwork;
    err = hipMalloc((void **)&dwork, sizeof(double) * lwork * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


// Try launching streams in multiple threads
#pragma omp parallel for
    for (int ib = 0; ib < nbatch; ib++) {
        double *dA = dA_batch + ib*nrow*ncol;
        double *db = db_batch + ib*nrow;
        hipsolverStatus_t serr;

        serr = hipsolverDnDgeqrf(cusolverH[ib], nrow, ncol, dA, nrow, dtau, dwork + lwork*ib, lwork, dinfo);
        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        // std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

        serr = hipsolverDnDormqr(cusolverH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork + lwork*ib, lwork, dinfo);
        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        //    std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

        double one(1.0);
        hipblasDtrsm(cublasH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

        hipMemcpyAsync(result_batch_ptr + ib*ncol, db, sizeof(double)*ncol, hipMemcpyDeviceToHost, streams[ib]);
    }


    //hipMemcpy(result_batch_ptr, db_batch, sizeof(double)*nrow*nbatch, hipMemcpyDeviceToHost);
    //for (int i = 0; i < nstream; i++)
    //    hipStreamSynchronize(streams[i]);

    hipDeviceSynchronize();



    hipFree(dA_batch);
    hipFree(dtau);
    hipFree(db_batch);
    hipFree(dinfo);
    hipFree(dwork);
}
