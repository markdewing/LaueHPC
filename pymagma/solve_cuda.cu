#include "hip/hip_runtime.h"

#include "solve_cuda.h"
#include "perf_info.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdexcept>


template<typename T>
void solve_cuda_QR(int nrow, int ncol, T* A_ptr, T* b_ptr, T* result_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    T* dA;
    err = hipMalloc((void **)&dA, sizeof(T) * nrow * ncol);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T *dtau;
    err = hipMalloc((void **)&dtau, sizeof(T) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    T* db;
    err = hipMalloc((void **)&db, sizeof(T) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");


    err = hipMemcpy(dA, A_ptr, sizeof(T)*nrow*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db, b_ptr, sizeof(T)*nrow, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");

    int lwork_geqrf;
    int lwork_ormqr;
    if constexpr(std::is_same<T,double>()) {
        hipsolverDnDgeqrf_bufferSize(cusolverH, nrow, ncol, dA, nrow, &lwork_geqrf);
        hipsolverDnDormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA, nrow, dtau, db, nrow, &lwork_ormqr);
    }
    if constexpr(std::is_same<T,float>()) {
        hipsolverDnSgeqrf_bufferSize(cusolverH, nrow, ncol, dA, nrow, &lwork_geqrf);
        hipsolverDnSormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA, nrow, dtau, db, nrow, &lwork_ormqr);
    }


    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    T *dwork;
    err = hipMalloc((void **)&dwork, sizeof(T) * lwork);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


    hipsolverStatus_t serr;
    if constexpr(std::is_same<T,double>())
        serr = hipsolverDnDgeqrf(cusolverH, nrow, ncol, dA, nrow, dtau, dwork, lwork, dinfo);
    if constexpr(std::is_same<T,float>())
        serr = hipsolverDnSgeqrf(cusolverH, nrow, ncol, dA, nrow, dtau, dwork, lwork, dinfo);

    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
     std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

    if constexpr(std::is_same<T,double>())
        serr = hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork, lwork, dinfo);
    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgeqrf failed");
    hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost);
    if (info != 0)
        std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

    T one(1.0);
    if constexpr(std::is_same<T,double>())
        hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

    hipMemcpy(result_ptr, db, sizeof(T)*ncol, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipFree(dA);
    hipFree(dtau);
    hipFree(db);
    hipFree(dinfo);
    hipFree(dwork);
}

template void solve_cuda_QR<double>(int nrow, int ncol, double* A_ptr, double* b_ptr, double* result_ptr, PerfInfo& perf);
template void solve_cuda_QR<float>(int nrow, int ncol, float* A_ptr, float* b_ptr, float* result_ptr, PerfInfo& perf);


template<typename T>
void solve_cuda_SVD(int nrow, int ncol, T* A_ptr, T* b_ptr, T* result_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    T* dA;
    err = hipMalloc((void **)&dA, sizeof(T) * nrow * ncol);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T* db;
    err = hipMalloc((void **)&db, sizeof(T) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");

    T* S = new T[min_mn];

    T* dS;
    err = hipMalloc((void **)&dS, sizeof(T) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dS");

    T* dU;
    err = hipMalloc((void **)&dU, sizeof(T) * nrow * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dU");

    T* dVT;
    err = hipMalloc((void **)&dVT, sizeof(T) * ncol * ncol);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dVT");



    err = hipMemcpy(dA, A_ptr, sizeof(T)*nrow*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db, b_ptr, sizeof(T)*nrow, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");


    int lwork_gesvd;
    if constexpr(std::is_same<T,double>())
        hipsolverDnDgesvd_bufferSize(cusolverH, nrow, ncol, &lwork_gesvd);
    if constexpr(std::is_same<T,float>())
        hipsolverDnSgesvd_bufferSize(cusolverH, nrow, ncol, &lwork_gesvd);

    int lwork = lwork_gesvd;

    printf("lwork size = %d\n",lwork);
    T *dwork;
    err = hipMalloc((void **)&dwork, sizeof(T) * lwork);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");

    char jobu('A');
    char jobvt('A');

    hipsolverStatus_t serr;
    if constexpr(std::is_same<T,double>())
        serr = hipsolverDnDgesvd(cusolverH, jobu, jobvt, nrow, ncol, dA, nrow, dS, dU, nrow, dVT, ncol, dwork, lwork, nullptr, dinfo);
    if constexpr(std::is_same<T,float>())
        serr = hipsolverDnSgesvd(cusolverH, jobu, jobvt, nrow, ncol, dA, nrow, dS, dU, nrow, dVT, ncol, dwork, lwork, nullptr, dinfo);

    if (serr != HIPSOLVER_STATUS_SUCCESS)
        throw std::runtime_error("hipsolverDnDgesvd failed"  + std::to_string(serr));

    err = hipMemcpy(S, dS, sizeof(T)*min_mn, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy S_ptr");

    for (int i = 0; i < ncol; i++) {
        S[i] = 1.0/S[i];
    }

    T* tmp_ptr  = new T[nrow];
    T* dtmp;
    err = hipMalloc((void **)&dtmp, sizeof(T) * nrow);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtmp");

    // u.T * b
    T one(1.0);
    T zero(0.0);
    int incx(1);
    if constexpr(std::is_same<T,double>())
        hipblasDgemv(cublasH, HIPBLAS_OP_T, nrow, nrow, &one, dU, nrow, db, incx, &zero, dtmp, incx);
    if constexpr(std::is_same<T,float>())
        hipblasSgemv(cublasH, HIPBLAS_OP_T, nrow, nrow, &one, dU, nrow, db, incx, &zero, dtmp, incx);

    err = hipMemcpy(tmp_ptr, dtmp, sizeof(T)*ncol, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy tmp_ptr to host");

    // S^-1 * (u.T * b)
    for (int i = 0; i < ncol; i++) {
        tmp_ptr[i] *= S[i];
    }

    err = hipMemcpy(dtmp, tmp_ptr, sizeof(T)*ncol, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy tmp_ptr to device");

    // v.T * (S^-1 * (u.T * b))
    if constexpr(std::is_same<T,double>())
        hipblasDgemv(cublasH, HIPBLAS_OP_T, ncol, ncol, &one, dVT, ncol, dtmp, incx, &zero, db, incx);
    if constexpr(std::is_same<T,float>())
        hipblasSgemv(cublasH, HIPBLAS_OP_T, ncol, ncol, &one, dVT, ncol, dtmp, incx, &zero, db, incx);

    hipMemcpy(result_ptr, db, sizeof(T)*ncol, hipMemcpyDeviceToHost);

    hipFree(dtmp);
    hipFree(dwork);
    hipFree(dVT);
    hipFree(dU);
    hipFree(dS);
    hipFree(db);
    hipFree(dA);

    hipsolverDnDestroy(cusolverH);
    hipblasDestroy(cublasH);

    delete[] S;
    delete[] tmp_ptr;
}

template void solve_cuda_SVD<double>(int nrow, int ncol, double* A_ptr, double* b_ptr, double* result_ptr, PerfInfo& perf);
template void solve_cuda_SVD<float>(int nrow, int ncol, float* A_ptr, float* b_ptr, float* result_ptr, PerfInfo& perf);

template<typename T>
void solve_batch_cuda_QR(int nrow, int ncol, int nbatch, T* A_batch_ptr, T* b_batch_ptr, T* result_batch_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    int nstream = nbatch;
    hipStream_t* streams = new hipStream_t[nstream];
    hipsolverHandle_t* cusolverH = new hipsolverHandle_t[nstream];
    hipblasHandle_t* cublasH = new hipblasHandle_t[nstream];
    for (int i = 0; i < nstream; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        //hipStreamCreate(&streams[i]);
        hipsolverDnCreate(&cusolverH[i]);
        hipsolverSetStream(cusolverH[i], streams[i]);
        hipblasCreate(&cublasH[i]);
        hipblasSetStream(cublasH[i], streams[i]);
    }

    int min_mn = nrow < ncol ? nrow : ncol;

    hipError_t err;
    T* dA_batch;
    err = hipMalloc((void **)&dA_batch, sizeof(T) * nrow * ncol * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T* dtau;
    err = hipMalloc((void **)&dtau, sizeof(T) * min_mn);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dtau");

    T* db_batch;
    err = hipMalloc((void **)&db_batch, sizeof(T) * nrow * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    int info;
    int *dinfo;
    err = hipMalloc((void **)&dinfo, sizeof(int));
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dinfo");

#if 0
    err = hipMemcpy(dA_batch, A_batch_ptr, sizeof(double)*nrow*ncol*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db_batch, b_batch_ptr, sizeof(double)*nrow*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");
#endif
#if 1
    for (int ib = 0; ib < nbatch; ib++)  {
        err = hipMemcpyAsync(dA_batch + ib*nrow*ncol, A_batch_ptr, sizeof(T)*nrow*ncol, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy dA");

        err = hipMemcpyAsync(db_batch + ib*nrow , b_batch_ptr, sizeof(T)*nrow, hipMemcpyHostToDevice, streams[ib]);
        if (err != hipSuccess)
            throw std::runtime_error("failed to copy db");
    }
#endif


    int lwork_geqrf;
    int lwork_ormqr;
    if constexpr(std::is_same<T,double>()) {
        hipsolverDnDgeqrf_bufferSize(cusolverH[0], nrow, ncol, dA_batch, nrow, &lwork_geqrf);
        hipsolverDnDormqr_bufferSize(cusolverH[0], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA_batch, nrow, dtau, db_batch, nrow, &lwork_ormqr);
    }

    if constexpr(std::is_same<T,float>()) {
        hipsolverDnSgeqrf_bufferSize(cusolverH[0], nrow, ncol, dA_batch, nrow, &lwork_geqrf);
        hipsolverDnSormqr_bufferSize(cusolverH[0], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, nrow, dA_batch, nrow, dtau, db_batch, nrow, &lwork_ormqr);
    }

    int lwork = std::max(lwork_geqrf, lwork_ormqr);

    printf("lwork size = %d\n",lwork);
    T* dwork;
    err = hipMalloc((void **)&dwork, sizeof(T) * lwork * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate dwork");


// Try launching streams in multiple threads
#pragma omp parallel for
    for (int ib = 0; ib < nbatch; ib++) {
        T* dA = dA_batch + ib*nrow*ncol;
        T* db = db_batch + ib*nrow;
        hipsolverStatus_t serr;

        if constexpr(std::is_same<T,double>())
            serr = hipsolverDnDgeqrf(cusolverH[ib], nrow, ncol, dA, nrow, dtau, dwork + lwork*ib, lwork, dinfo);
        if constexpr(std::is_same<T,float>())
            serr = hipsolverDnSgeqrf(cusolverH[ib], nrow, ncol, dA, nrow, dtau, dwork + lwork*ib, lwork, dinfo);

        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        // std::runtime_error(std::string("dgeqrf info = ") + std::to_string(info));

        if constexpr(std::is_same<T,double>())
            serr = hipsolverDnDormqr(cusolverH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork + lwork*ib, lwork, dinfo);
        if constexpr(std::is_same<T,float>())
            serr = hipsolverDnSormqr(cusolverH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, nrow, 1, ncol, dA, nrow, dtau, db, nrow, dwork + lwork*ib, lwork, dinfo);
        if (serr != HIPSOLVER_STATUS_SUCCESS)
            throw std::runtime_error("hipsolverDnDgeqrf failed");
        //hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, streams[ib]);
        //if (info != 0)
        //    std::runtime_error(std::string("dormqr info = ") + std::to_string(info));

        T one(1.0);
        if constexpr(std::is_same<T,double>())
            hipblasDtrsm(cublasH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);
        if constexpr(std::is_same<T,float>())
            hipblasStrsm(cublasH[ib], HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, ncol, 1, &one, dA, nrow, db, nrow);

        hipMemcpyAsync(result_batch_ptr + ib*ncol, db, sizeof(T)*ncol, hipMemcpyDeviceToHost, streams[ib]);
    }


    //hipMemcpy(result_batch_ptr, db_batch, sizeof(double)*nrow*nbatch, hipMemcpyDeviceToHost);
    //for (int i = 0; i < nstream; i++)
    //    hipStreamSynchronize(streams[i]);

    hipDeviceSynchronize();



    hipFree(dA_batch);
    hipFree(dtau);
    hipFree(db_batch);
    hipFree(dinfo);
    hipFree(dwork);
}

template void solve_batch_cuda_QR<double>(int nrow, int ncol, int nbatch, double* A_batch_ptr, double* b_batch_ptr, double* result_batch_ptr, PerfInfo& perf);
template void solve_batch_cuda_QR<float>(int nrow, int ncol, int nbatch, float* A_batch_ptr, float* b_batch_ptr, float* result_batch_ptr, PerfInfo& perf);

template<typename T>
void solve_batch_cuda_LS(int nrow, int ncol, int nbatch, T* A_batch_ptr, T* b_batch_ptr, T* result_batch_ptr, PerfInfo& perf)
{
    RecordElapsed recordElapsed(perf);

    hipblasHandle_t cublasH = nullptr;
    hipblasCreate(&cublasH);

    hipError_t err;
    T* dA_batch;
    err = hipMalloc((void **)&dA_batch, sizeof(T) * nrow * ncol * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T** dA_ptrs;
    err = hipMalloc((void ***)&dA_ptrs, sizeof(T*) * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dA_ptrs: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T** A_ptrs = new T*[nbatch];
    for (int i = 0; i < nbatch; i++) {
        A_ptrs[i] = dA_batch + i*nrow*ncol;
    }

    err = hipMemcpy(dA_ptrs, A_ptrs, sizeof(T*)*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    T* db_batch;
    err = hipMalloc((void **)&db_batch, sizeof(T) * nrow * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error("failed to allocate db");

    T** db_ptrs;
    err = hipMalloc((void ***)&db_ptrs, sizeof(T*) * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate db_ptrs: ") + hipGetErrorName(err) + hipGetErrorString(err));

    T** b_ptrs = new T*[nbatch];
    for (int i = 0; i < nbatch; i++) {
        b_ptrs[i] = db_batch + i*nrow;
    }

    err = hipMemcpy(db_ptrs, b_ptrs, sizeof(T*)*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy b_ptrs");

    err = hipMemcpy(dA_batch, A_batch_ptr, sizeof(T)*nrow*ncol*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy dA");

    err = hipMemcpy(db_batch, b_batch_ptr, sizeof(T)*nrow*nbatch, hipMemcpyHostToDevice);
    if (err != hipSuccess)
        throw std::runtime_error("failed to copy db");

    int* dinfos;
    err = hipMalloc((void **)&dinfos, sizeof(int) * nbatch);
    if (err != hipSuccess)
        throw std::runtime_error(std::string("failed to allocate dinfos: ") + hipGetErrorName(err) + hipGetErrorString(err));

    int info;
    int nrhs = 1;
    if constexpr(std::is_same<T,double>())
        hipblasDgelsBatched(cublasH, HIPBLAS_OP_N, nrow, ncol, nrhs, dA_ptrs, nrow, db_ptrs, nrow, &info, dinfos, nbatch);
    if constexpr(std::is_same<T,float>())
        hipblasSgelsBatched(cublasH, HIPBLAS_OP_N, nrow, ncol, nrhs, dA_ptrs, nrow, db_ptrs, nrow, &info, dinfos, nbatch);

    if (info != 0)
        throw std::runtime_error("hipblasDgelsBatched info not zero : " + std::to_string(info));
    for (int ib = 0; ib < nbatch; ib++)
    {
        hipMemcpy(result_batch_ptr + ib*ncol, db_batch, sizeof(T)*ncol, hipMemcpyDeviceToHost);
    }
}

template void solve_batch_cuda_LS<double>(int nrow, int ncol, int nbatch, double* A_batch_ptr, double* b_batch_ptr, double* result_batch_ptr, PerfInfo& perf);
template void solve_batch_cuda_LS<float>(int nrow, int ncol, int nbatch, float* A_batch_ptr, float* b_batch_ptr, float* result_batch_ptr, PerfInfo& perf);
